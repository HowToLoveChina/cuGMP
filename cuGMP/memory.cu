#include <stdio.h>

#include "cuGMP.h"

#include "hip/hip_runtime.h"
#include ""
#include "include/hip/hip_runtime_api.h"

void allocate_memory(mpz_ptr x, size_t limbs, size_t size)
{
	__dev_mpz_struct dev_x;
	checkCudaErrors(hipMemcpy(&dev_x, x->_dev_mp_struct, sizeof(__dev_mpz_struct), hipMemcpyDeviceToHost));

	if (x->_mp_alloc < limbs)
	{
		hipFree(dev_x._mp_d);
		free(x->_mp_d);
		checkCudaErrors(hipMalloc((void**)&dev_x._mp_d, sizeof(mp_limb_t) * limbs));
		x->_mp_d = (mp_limb_t *)malloc(limbs * sizeof(x->_mp_d));
		x->_mp_alloc = (int)limbs;
	}
	x->_mp_size = (int)size;

	dev_x._mp_size = x->_mp_size;
	dev_x._mp_alloc = x->_mp_alloc;
	checkCudaErrors(hipMemcpy(x->_dev_mp_struct, &dev_x, sizeof(__dev_mpz_struct), hipMemcpyHostToDevice));
}

__dev_mpz_struct copy_operand(mpz_ptr x)
{
	__dev_mpz_struct dev_x;
	checkCudaErrors(hipMemcpy(&dev_x, x->_dev_mp_struct, sizeof(__dev_mpz_struct), hipMemcpyDeviceToHost));
	return dev_x;
}

void copy_operand(mpz_ptr x, __dev_mpz_struct dev_x)
{
	checkCudaErrors(hipMemcpy(x->_dev_mp_struct, &dev_x, sizeof(__dev_mpz_struct), hipMemcpyHostToDevice));
}

void copy_operand_data_without_limbs(mpz_ptr x, MemcpyDirection direction)
{
	__dev_mpz_struct dev_x;

	switch (direction)
	{
	case MemcpyDirection::memcpyHostToDevice:
		// To preserve device pointers, it is vital to first copy actual struct from device, before altering it.
		checkCudaErrors(hipMemcpy(&dev_x, x->_dev_mp_struct, sizeof(__dev_mpz_struct), hipMemcpyDeviceToHost));
		dev_x._mp_size = x->_mp_size;
		checkCudaErrors(hipMemcpy(x->_dev_mp_struct, &dev_x, sizeof(__dev_mpz_struct), hipMemcpyHostToDevice));
		break;
	case MemcpyDirection::memcpyDeviceToHost:
		checkCudaErrors(hipMemcpy(&dev_x, x->_dev_mp_struct, sizeof(__dev_mpz_struct), hipMemcpyDeviceToHost));
		x->_mp_size = dev_x._mp_size;
		break;
	default:
		printf("Unsupported direction in copy_operand_data, skipping\n");
		break;
	}
}

void copy_operand_data_with_limbs(mpz_ptr x, MemcpyDirection direction)
{
	__dev_mpz_struct dev_x;

	switch (direction)
	{
	case MemcpyDirection::memcpyHostToDevice:
		checkCudaErrors(hipMemcpy(&dev_x, x->_dev_mp_struct, sizeof(__dev_mpz_struct), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(dev_x._mp_d, x->_mp_d, sizeof(mp_limb_t) * ABS(x->_mp_size), hipMemcpyHostToDevice));
		break;
	case MemcpyDirection::memcpyDeviceToHost:
		checkCudaErrors(hipMemcpy(&dev_x, x->_dev_mp_struct, sizeof(__dev_mpz_struct), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(x->_mp_d, dev_x._mp_d, sizeof(mp_limb_t) * ABS(dev_x._mp_size), hipMemcpyDeviceToHost));
		break;
	default:
		printf("Unsupported direction in copy_operand_data, skipping\n");
		break;
	}
}
