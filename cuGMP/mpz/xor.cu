#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "../include/hip/hip_runtime_api.h"

#include "../cuGMP.h"

__global__ void xorKernel(__dev_mpz_struct *res, const __dev_mpz_struct *a, const __dev_mpz_struct *b, unsigned int size)
{
	unsigned int i = THREAD_ID;

	if (i < size)
	{
		if (i >= a->_mp_size)
		{
			res->_mp_d[i] = b->_mp_d[i];
		}
		else if (i >= b->_mp_size)
		{
			res->_mp_d[i] = a->_mp_d[i];
		}
		else
		{
			res->_mp_d[i] = a->_mp_d[i] ^ b->_mp_d[i];
		}
	}
}

void mpz_xor(mpz_ptr res, mpz_ptr a, mpz_ptr b)
{
	unsigned int size = MAX(ABS(a->_mp_size), ABS(b->_mp_size));
	allocate_memory(res, size, size);

#ifdef KERNEL_PRINT
	printf("xorKernel <<<%d, %d>>>\n", size / BLOCK_SIZE + 1, BLOCK_SIZE);
#endif
	xorKernel << <size / BLOCK_SIZE + 1, BLOCK_SIZE >> >(res->_dev_mp_struct, a->_dev_mp_struct, b->_dev_mp_struct, size);
	getLastCudaError("Kernel execution failed: [ xorKernel ]");
#ifdef EXPLICIT_SYNCHRONIZATION
	checkCudaErrors(hipDeviceSynchronize());
#endif
}