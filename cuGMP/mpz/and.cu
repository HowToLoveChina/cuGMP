#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "../include/hip/hip_runtime_api.h"

#include "../cuGMP.h"

__global__ void andKernel(__dev_mpz_struct *res, const __dev_mpz_struct *a, const __dev_mpz_struct *b, unsigned int size)
{
	unsigned int i = THREAD_ID;

	if (i < size)
	{
		res->_mp_d[i] = a->_mp_d[i] & b->_mp_d[i];
	}
}

void mpz_and(mpz_ptr res, mpz_ptr a, mpz_ptr b)
{
	unsigned int size = MIN(ABS(a->_mp_size), ABS(b->_mp_size));
	allocate_memory(res, size, size);

#ifdef KERNEL_PRINT
	printf("andKernel <<<%d, %d>>>\n", size / BLOCK_SIZE + 1, BLOCK_SIZE);
#endif
	andKernel << <size / BLOCK_SIZE + 1, BLOCK_SIZE >> >(res->_dev_mp_struct, a->_dev_mp_struct, b->_dev_mp_struct, size);
	getLastCudaError("Kernel execution failed: [ andKernel ]");
#ifdef EXPLICIT_SYNCHRONIZATION
	checkCudaErrors(hipDeviceSynchronize());
#endif
}
